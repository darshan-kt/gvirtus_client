
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>

int main() {
    // Matrix A (2x3)
    float A[6] = {1, 2, 3, 4, 5, 6};
    // Matrix B (3x2)
    float B[6] = {7, 8, 9, 10, 11, 12};
    // Matrix C (2x2), the result
    float C[4] = {0};

    float *d_A, *d_B, *d_C;
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Allocate device memory
    hipMalloc((void **)&d_A, 6 * sizeof(float));
    hipMalloc((void **)&d_B, 6 * sizeof(float));
    hipMalloc((void **)&d_C, 4 * sizeof(float));

    // Copy matrices from host to device
    hipMemcpy(d_A, A, 6 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, 6 * sizeof(float), hipMemcpyHostToDevice);

    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Perform matrix multiplication: C = alpha * A * B + beta * C
    // A: 2x3, B: 3x2, C: 2x2
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 2, 2, 3, &alpha, d_A, 2, d_B, 3, &beta, d_C, 2);

    // Copy result back to host
    hipMemcpy(C, d_C, 4 * sizeof(float), hipMemcpyDeviceToHost);

    // Print result
    std::cout << "Matrix C: ";
    for (int i = 0; i < 4; i++) {
        std::cout << C[i] << " ";
    }
    std::cout << std::endl;

    // Free resources
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

